#include "hip/hip_runtime.h"
#include "cp.h"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include<vector>
using namespace std;

inline void check(hipError_t err, const char* context) 
{
    if (err != hipSuccess) 
    {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)
inline int static divup(int a, int b) 
{
    return (a + b - 1)/b;
}

inline int static roundup(int a, int b) 
{
    return divup(a, b) * b;
}

__global__ void mykernel(int nx, int ny, const float * data, float * result)
{
    int ia = threadIdx.x;
    int ja = threadIdx.y;
    int ic = blockIdx.x;
    int jc = blockIdx.y;

    if( ic < jc )
    {
        return;
    }
    float v[ 8 ][ 8 ];
    for( int ib = 0; ib < 8; ++ib )
    {
        for( int jb = 0; jb < 8; ++jb )
        {
            v[ ib ][ jb ] = 0;
        }
    }
    for( int k = 0; k < nx; ++k )
    {
        float x[ 8 ];
        float y[ 8 ];
        for( int ib = 0; ib < 8; ++ ib )
        {
            int i = ic * 64 + ib * 8 + ia;            
            x[ ib ] = data[ ny * k + i ];
            for( int jb = 0; jb < 8; ++ jb )
            {
                int j = jc * 64 + jb * 8 + ja;
                y[ jb ] = data[ ny * k + j ];
            }
            
        }       
        for( int ib = 0; ib < 8; ++ ib )
        {
            for( int jb = 0; jb < 8; ++jb )
            {
                v[ ib ][ jb ] += x[ ib ] * y[ jb ];
            }
        }
    }
    for( int ib = 0; ib < 8; ++ib )
    {
        for( int jb = 0; jb < 8; ++jb )
        {
            int i = ic * 64 + ib * 8 + ia;
            int j = jc * 64 + jb * 8 + ja;
            if( i < ny && j < ny )
            {
                result[ ny * j + i ] = v[ ib ][ jb ];
            }
        }
    }
}

__global__ void myppkernel( int nx, int ny, const float * data, float * data_copy )
{
    int y = threadIdx.x + blockIdx.x * blockDim.x ;    
    if( y >= ny )
    {
        return;
    }
    float s = 0.0;
    float t = 0.0;
    for( int x = 0; x < nx; ++x )
    {
        s += data[ x + y * nx ];
    }
    s = s / (float)nx;
    for( int x = 0; x < nx; ++x )
    {
        data_copy[ y + x * ny ] = data[ x + y*nx ] -s;
        t +=  data_copy[ y + x * ny ]* data_copy[ y + x * ny ];
    }
    t = sqrt( t );
    for( int x = 0; x < nx; x ++ )
    {
        data_copy[ y + x * ny ] = data_copy[ y + x * ny ]/t;
    }
}

void correlate(int ny, int nx, const float* data, float* result) 
{
    //preprocess data into data_copy
    int ny_64 = roundup( ny, 64 );

    //Allocate memory and copy data to GPU
    float * dGPU = NULL;//data for GPU
    CHECK(hipMalloc((void**)&dGPU, ny_64 * ny_64 * sizeof(float)));
    
    float *dpGPU = NULL;//original data for GPU preproc
    CHECK(hipMalloc((void**)&dpGPU, nx * ny * sizeof(float)));

    float * rGPU = NULL;//result for GPU
    CHECK(hipMalloc((void**)&rGPU, ny * ny * sizeof(float)));
    
    CHECK(hipMemcpy(dpGPU, data, nx * ny * sizeof(float), hipMemcpyHostToDevice));

    //Run kernel
    {
        dim3 dimBlock(64);
        dim3 dimGrid( divup(ny,64) );
        myppkernel<<<dimGrid, dimBlock>>>( nx, ny, dpGPU, dGPU );
    }
    //Run kernel
    {
        dim3 dimBlock(8, 8);
        dim3 dimGrid( ny_64/64, ny_64/64 );
        mykernel<<<dimGrid, dimBlock>>>( nx, ny, dGPU, rGPU);
        CHECK(hipGetLastError());
    }
    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, rGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost ));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}
