#include "hip/hip_runtime.h"
#include "is.h"
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

inline void check(hipError_t err, const char* context) 
{
    if (err != hipSuccess) 
    {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)
inline int static divup(int a, int b) 
{
    return (a + b - 1)/b;
}

__global__ void mykernel(int nx, int ny, const float * data, float * result)
{
    int len_x = threadIdx.x + blockIdx.x * blockDim.x;
    int len_y = threadIdx.y + blockIdx.y * blockDim.y;

    if( len_x == 0 || len_y ==0 || len_x > nx || len_y > ny || ( len_x == nx && len_y == ny ) )
    {
        return;
    }
    const int sum_nx = nx + 1;
    const int sum_ny = ny + 1;
    const int X = len_x * len_y;
    const int Y = nx * ny - X;
    const float X_inv = 1/(float)X;
    const float Y_inv = 1/(float)Y;
    const float sy = data[( sum_nx - 1 )  + sum_nx * ( sum_ny -1 )];
    float h_max = 0;
    for( int pos_y = 1; pos_y <= sum_ny - len_y; ++pos_y )
    {
        for( int pos_x = 1; pos_x <= sum_nx - len_x; pos_x++ )
        {

            int y1 = pos_y + len_y - 1;
            int x1 = pos_x + len_x - 1;
            float s1 = data[x1 + sum_nx * y1];
            float s2 = data[x1 + sum_nx * ( pos_y - 1 )];
            float s3 = data[( pos_x - 1 ) + sum_nx * y1];
            float s4 = data[ ( pos_x - 1 ) + sum_nx * ( pos_y - 1) ];
            float vx = s1 - s2 - s3 + s4;
            float vy = sy - vx;
            float a = (vx * vx) * X_inv + vy * vy * Y_inv;
            if( a > h_max )
            {
                h_max = a;
            }
        }
    }
    result[ len_x + len_y * sum_nx ] = h_max;
}

Result segment(int ny, int nx, const float* data) 
{
    Result result { 0, 0, 0, 0, {0, 0, 0}, {0, 0, 0} };
    
    const int sum_nx = nx + 1;
    const int sum_ny = ny + 1;
    float * s = ( float * )calloc( sum_nx * sum_ny , sizeof(float)  ) ;

    for(int x = 0; x < sum_nx; x++ )
    {
        s[x] = 0;
    }
    for(int y = 0; y < sum_ny; y++ )
    {
        s[sum_nx * y] = 0;
    }

    s[ 1 + sum_nx] = data[0];


    for( int x = 3; x < 3 * nx; x += 3 )
    {
        s[ 1 + x/3 + sum_nx ] = data[ x ] + s[ 1 + x/3 - 1 + sum_nx ];
    }
    for ( int y = 2; y < sum_ny; ++y )
    {
        for ( int x = 1; x < sum_nx; ++x )
        {
            int ind  =  x + sum_nx * y;         
            int ind3 =  x + sum_nx * ( y - 1 ); 
            int ind2 = ( x -1 ) + sum_nx * y;   
            int ind4 = ( x -1 ) + sum_nx * (y-1);
            s[ ind ] = data[ 3 * (x-1) + 3 * nx * (y-1)  ] + s[ ind2 ] - s[ ind4 ] + s[ ind3 ];
            
        }
    }

    float * dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, sum_nx * sum_ny * sizeof(float)));
    CHECK(hipMemcpy(dGPU, s, sum_nx * sum_ny * sizeof(float), hipMemcpyHostToDevice));

    float * rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, sum_nx * sum_ny * sizeof(float)));

    
    {
        dim3 dimBlock(8, 8);
        dim3 dimGrid( divup(nx,dimBlock.x), divup(ny,dimBlock.y) );
        mykernel<<<dimGrid, dimBlock>>>( nx, ny, dGPU, rGPU);
        CHECK(hipGetLastError());
    }
    float * result_array = (float * )malloc( sizeof(float) * sum_nx * sum_ny ) ; 
    CHECK(hipMemcpy(result_array, rGPU, sum_nx * sum_ny * sizeof(float), hipMemcpyDeviceToHost ));

    float h_max = 0;
    int best_X = 0, best_Y = 0;
    for( int len_y = 1; len_y < sum_ny; ++len_y )
    {
        for( int len_x = 1; len_x < sum_nx; ++len_x )
        {
            if( result_array[ len_x + sum_nx * len_y ] > h_max )
            {
                h_max = result_array[ len_x + sum_nx * len_y ];
                best_X = len_x;
                best_Y = len_y;
            }
        }
    }


    const int X = best_X  * best_Y;
    const int Y = nx * ny - X;
    const float X_inv = 1/(float)X;
    const float Y_inv = 1/(float)Y;
    const float sy = s[ ( sum_nx - 1 )  + sum_nx * ( sum_ny - 1 )];

    h_max = 0;
    for( int pos_y = 1; pos_y <= sum_ny - best_Y; pos_y++ )
    {
        for( int pos_x = 1; pos_x <= sum_nx - best_X; pos_x++ )
        {

            int y1 = pos_y + best_Y - 1;
            int x1 = pos_x + best_X - 1;
            float s1 = s[ x1 + sum_nx * y1 ];
            float s2 = s[ x1 + sum_nx * ( pos_y - 1 ) ];
            float s3 = s[ ( pos_x - 1 ) + sum_nx * y1 ];
            float s4 = s[ ( pos_x - 1 ) + sum_nx * ( pos_y - 1) ];
            float vx = s1 - s2 - s3 + s4;
            float vy = sy - vx;
            float a = (vx * vx) * X_inv + (vy * vy) * Y_inv;

            if( a > h_max )
            {
                vx *= X_inv;
                vy *= Y_inv;
                result.x0 = pos_x-1;
                result.x1 = x1;
                result.y0 = pos_y-1;
                result.y1 = y1;
                result.outer[ 0 ] = (vy);
                result.outer[ 1 ] = (vy);
                result.outer[ 2 ] = (vy);
                result.inner[ 0 ] = (vx);
                result.inner[ 1 ] = (vx);
                result.inner[ 2 ] = (vx);
                h_max = a;
            }
        }
    }

    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
    std::free(s);
	std::free(result_array);
 	return result;
}
